#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <assert.h>
#include <malloc.h>
#include <math.h>
#include <stdlib.h>
// CUDA runtime
#include <hip/hip_runtime.h>
// Helper functions and utilities to work with CUDA
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"

#ifndef BLOCKSIZE
#define BLOCKSIZE		32     // number of threads per block
#endif

const int Size = 32769;

//Function Prototypes
double square (int);

__global__ void AutoCorrelate(float *dA, float *dSum){
    int gid  = blockIdx.x*blockDim.x + threadIdx.x;
    int shift = gid;
    float sum = 0.;

    for( int i = 0; i < Size; i++ )
    {
        sum += (float)dA[i] * dA[i + shift];
    }
    dSum[shift] = sum;
}

int main( int argc, char* argv[ ] ){
    FILE *fp = fopen( "signal.txt", "r" );
    if( fp == NULL ){
        fprintf( stderr, "Cannot open file 'signal.txt'\n" );
        exit( 1 );
    }
    int Size;
    fscanf( fp, "%d", &Size );
    float *hA =     new float[ 2*Size ];
    float *hSum  = new float[ 1*Size ];

    for( int i = 0; i < Size; i++ ){
        fscanf( fp, "%f", &hA[i] );
        hA[i+Size] = hA[i];		// duplicate the array
    }
    fclose( fp );

	int dev = findCudaDevice(argc, (const char **)argv);

	// allocate host memory:
	float *dA, *dSum;

	dim3 dimArray(2*Size, 1, 1);
	dim3 dimSum(1*Size, 1, 1);

	hipError_t status;
	status = hipMalloc((void **)(&dA), 2*Size*sizeof(float));
	checkCudaErrors(status);

	status = hipMalloc((void **)(&dSum), 1*Size*sizeof(float));
	checkCudaErrors(status);

	// copy host memory to the device:
	status = hipMemcpy( dA, hA, 2*Size*sizeof(float), hipMemcpyHostToDevice );
	checkCudaErrors( status );

    //Nothing to copy ?
	// status = hipMemcpy( dSum, hSum, NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
	// checkCudaErrors( status );

	// setup the execution parameters:
	dim3 threads(BLOCKSIZE, 1, 1 );
	dim3 grid(Size/threads.x, 1, 1 );

	// create and start timer
	hipDeviceSynchronize( );

	// allocate CUDA events that we'll use for timing:
	hipEvent_t start, stop;
	status = hipEventCreate( &start );
	checkCudaErrors( status );
	status = hipEventCreate( &stop );
	checkCudaErrors( status );

	// record the start event:
	status = hipEventRecord( start, NULL );
	checkCudaErrors( status );

	// execute the kernel:
	AutoCorrelate<<< grid, threads >>>(dA, dSum);

	// record the stop event:
	status = hipEventRecord( stop, NULL );
	checkCudaErrors( status );

	// wait for the stop event to complete:

	status = hipEventSynchronize( stop );
	checkCudaErrors( status );

	float msecTotal = 0.0f;
	status = hipEventElapsedTime( &msecTotal, start, stop );
	checkCudaErrors( status );

	// compute and print the performance
	double secondsTotal = 0.001 * (double)msecTotal;
	double megaCorsPerSecond = square(Size) / secondsTotal / 1000000.;
	fprintf( stderr, "Block size = %d, MegaTrials/Second = %10.4lf\n", BLOCKSIZE, megaCorsPerSecond );

	// copy result from the device to the host:
	status = hipMemcpy( hSum, dSum, Size *sizeof(float), hipMemcpyDeviceToHost );
	checkCudaErrors( status );

    if (BLOCKSIZE == 32) {
        FILE *fw = fopen( "CUDA.txt", "a" );
        for( int i = 1; i < 513; i++ ){
            fprintf( fw, "%d %f \n", i, hSum[i]);
        }
        fclose( fw );
    }
	// clean up memory:
	delete [ ] hA;
	delete [ ] hSum;

    status = hipFree( dA );
    checkCudaErrors(status);
	status = hipFree( dSum );
    checkCudaErrors(status);
	return 0;
}

double square (int x){
    return x*x;
}
